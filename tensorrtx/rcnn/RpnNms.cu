#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/gather.h>
#include <thrust/system/cuda/detail/cub/device/device_radix_sort.cuh>

#include <algorithm>
#include <iostream>
#include <stdexcept>
#include <cstdint>
#include <vector>
#include <cmath>

#include "RpnNmsPlugin.h"
#include "./cuda_utils.h"

namespace nvinfer1 {

    __global__ void rpn_nms_kernel(
        const float threshold, const int num_detections,
        const int *indices, float *scores, const float4 *boxes) {
        // Go through detections by descending score
        for (int m = 0; m < num_detections; m++) {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            if (i < num_detections && m < i && scores[m] > -FLT_MAX) {
                int idx = indices[i];
                int max_idx = indices[m];

                float4 ibox = boxes[idx];
                float4 mbox = boxes[max_idx];
                float x1 = max(ibox.x, mbox.x);
                float y1 = max(ibox.y, mbox.y);
                float x2 = min(ibox.z, mbox.z);
                float y2 = min(ibox.w, mbox.w);
                float w = max(0.0f, x2 - x1);
                float h = max(0.0f, y2 - y1);
                float iarea = (ibox.z - ibox.x) * (ibox.w - ibox.y);
                float marea = (mbox.z - mbox.x) * (mbox.w - mbox.y);
                float inter = w * h;
                float overlap = inter / (iarea + marea - inter);
                if (overlap > threshold) {
                    scores[i] = -FLT_MAX;
                }
            }

            // Sync discarded detections
            __syncthreads();
        }
    }

    int rpnNms(int batch_size,
        const void *const *inputs, void **outputs,
        size_t pre_nms_topk, int post_nms_topk, float nms_thresh,
        void *workspace, size_t workspace_size, hipStream_t stream) {
        if (!workspace || !workspace_size) {
            // Return required scratch space size cub style
            workspace_size += get_size_aligned<int>(pre_nms_topk);   // indices
            workspace_size += get_size_aligned<int>(pre_nms_topk);   // indices_sorted
            workspace_size += get_size_aligned<float>(pre_nms_topk);  // scores
            workspace_size += get_size_aligned<float>(pre_nms_topk);  // scores_sorted

            size_t temp_size_sort = 0;
            thrust::cuda_cub::hipcub::DeviceRadixSort::SortPairsDescending(
                static_cast<void*>(nullptr), temp_size_sort,
                static_cast<float*>(nullptr),
                static_cast<float*>(nullptr),
                static_cast<int*>(nullptr),
                static_cast<int*>(nullptr), pre_nms_topk);
            workspace_size += temp_size_sort;

            return workspace_size;
        }

        auto on_stream = thrust::cuda::par.on(stream);

        auto indices = get_next_ptr<int>(pre_nms_topk, workspace, workspace_size);
        std::vector<int> indices_h(pre_nms_topk);
        for (int i = 0; i < pre_nms_topk; i++)
            indices_h[i] = i;
        hipMemcpyAsync(indices, indices_h.data(), pre_nms_topk * sizeof * indices, hipMemcpyHostToDevice, stream);
        auto indices_sorted = get_next_ptr<int>(pre_nms_topk, workspace, workspace_size);
        auto scores = get_next_ptr<float>(pre_nms_topk, workspace, workspace_size);
        auto scores_sorted = get_next_ptr<float>(pre_nms_topk, workspace, workspace_size);

        for (int batch = 0; batch < batch_size; batch++) {
            auto in_scores = static_cast<const float *>(inputs[0]) + batch * pre_nms_topk;
            auto in_boxes = static_cast<const float4 *>(inputs[1]) + batch * pre_nms_topk;

            auto out_boxes = static_cast<float4 *>(outputs[0]) + batch * post_nms_topk;

            int num_detections = pre_nms_topk;
            thrust::cuda_cub::hipcub::DeviceRadixSort::SortPairsDescending(workspace, workspace_size,
                in_scores, scores_sorted, indices, indices_sorted, num_detections, 0,
                sizeof(*scores_sorted) * 8, stream);

            // Launch actual NMS kernel - 1 block with each thread handling n detections
            // TODO: different device has differnet max threads
            const int max_threads = 1024;
            int num_per_thread = ceil(static_cast<float>(num_detections) / max_threads);
            rpn_nms_kernel << <num_per_thread, max_threads, 0, stream >> > (nms_thresh, num_detections,
                indices_sorted, scores_sorted, in_boxes);

            // Re-sort with updated scores
            thrust::cuda_cub::hipcub::DeviceRadixSort::SortPairsDescending(workspace, workspace_size,
                scores_sorted, scores, indices_sorted, indices, num_detections, 0, sizeof(*scores_sorted) * 8, stream);

            // Gather filtered scores, boxes, classes
            num_detections = min(post_nms_topk, num_detections);
            thrust::gather(on_stream, indices, indices + num_detections, in_boxes, out_boxes);
        }

        return 0;
    }
}  // namespace nvinfer1
