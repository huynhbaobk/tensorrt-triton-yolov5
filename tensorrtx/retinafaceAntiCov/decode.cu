#include "hip/hip_runtime.h"
#include "decode.h"
#include "stdio.h"

namespace nvinfer1
{
    DecodePlugin::DecodePlugin()
    {
    }

    DecodePlugin::~DecodePlugin()
    {
    }

    // create the plugin at runtime from a byte stream
    DecodePlugin::DecodePlugin(const void* data, size_t length)
    {
    }

    void DecodePlugin::serialize(void* buffer) const
    {
    }

    size_t DecodePlugin::getSerializationSize() const
    {  
        return 0;
    }

    int DecodePlugin::initialize()
    { 
        return 0;
    }

    Dims DecodePlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        //output the result to channel
        int totalCount = 0;
        totalCount += decodeplugin::INPUT_H / 8 * decodeplugin::INPUT_W / 8 * 2 * sizeof(decodeplugin::Detection) / sizeof(float);
        totalCount += decodeplugin::INPUT_H / 16 * decodeplugin::INPUT_W / 16 * 2 * sizeof(decodeplugin::Detection) / sizeof(float);
        totalCount += decodeplugin::INPUT_H / 32 * decodeplugin::INPUT_W / 32 * 2 * sizeof(decodeplugin::Detection) / sizeof(float);

        return Dims3(totalCount + 1, 1, 1);
    }

    // Set plugin namespace
    void DecodePlugin::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* DecodePlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType DecodePlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool DecodePlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool DecodePlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void DecodePlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void DecodePlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void DecodePlugin::detachFromContext() {}

    const char* DecodePlugin::getPluginType() const
    {
        return "Decode_TRT";
    }

    const char* DecodePlugin::getPluginVersion() const
    {
        return "1";
    }

    void DecodePlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* DecodePlugin::clone() const
    {
        DecodePlugin *p = new DecodePlugin();
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data){ return 1./(1. + expf(-data)); };

    __global__ void CalDetection(const float *input, float *output, int num_elem, int step, int anchor) {

        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= num_elem) return;

        int h = decodeplugin::INPUT_H / step;
        int w = decodeplugin::INPUT_W / step;
        int y = idx / w;
        int x = idx % w;
        const float *cls_reg = &input[2 * num_elem];
        const float *bbox_reg = &input[4 * num_elem];
        const float *lmk_reg = &input[12 * num_elem];
        const float *mask_reg = &input[36 * num_elem];

        for (int k = 0; k < 2; ++k) {
            float conf = cls_reg[idx + k * num_elem];
            if (conf < 0.5) continue;

            float *res_count = output;
            int count = (int)atomicAdd(res_count, 1);
            char* data = (char *)res_count + sizeof(float) + count * sizeof(decodeplugin::Detection);
            decodeplugin::Detection* det = (decodeplugin::Detection*)(data);

            float prior[4];
            prior[0] = 7.5 + (float)(x * step);
            prior[1] = 7.5 + (float)(y * step);
            prior[2] = anchor * 2 / (k + 1);
            prior[3] = prior[2];

            //Location
            det->bbox[0] = prior[0] + bbox_reg[idx + k * num_elem * 4] * prior[2];
            det->bbox[1] = prior[1] + bbox_reg[idx + k * num_elem * 4 + num_elem] * prior[3];
            det->bbox[2] = prior[2] * expf(bbox_reg[idx + k * num_elem * 4 + num_elem * 2]);
            det->bbox[3] = prior[3] * expf(bbox_reg[idx + k * num_elem * 4 + num_elem * 3]);
            det->bbox[0] -= (det->bbox[2] - 1) / 2;
            det->bbox[1] -= (det->bbox[3] - 1) / 2;
            det->bbox[2] += det->bbox[0];
            det->bbox[3] += det->bbox[1];
            det->class_confidence = conf;
            for (int i = 0; i < 10; i += 2) {
                det->landmark[i] = prior[0] + lmk_reg[idx + k * num_elem * 10 + num_elem * i] * 0.2 * prior[2];
                det->landmark[i+1] = prior[1] + lmk_reg[idx + k * num_elem * 10 + num_elem * (i + 1)] * 0.2 * prior[3];
            }
            det->mask_confidence = mask_reg[idx + k * num_elem];;
        }
    }

    void DecodePlugin::forwardGpu(const float *const * inputs, float * output, hipStream_t stream, int batchSize) 
    {
        int num_elem = 0;
        int base_step = 8;
        int base_anchor = 16;
        int thread_count;
        hipMemset(output, 0, sizeof(float));
        for (unsigned int i = 0; i < 3; ++i)
        {
            num_elem = decodeplugin::INPUT_H / base_step * decodeplugin::INPUT_W / base_step;
            thread_count = (num_elem < thread_count_) ? num_elem : thread_count_;
            CalDetection<<< (num_elem + thread_count - 1) / thread_count, thread_count>>>
                (inputs[i], output, num_elem, base_step, base_anchor);
            base_step *= 2;
            base_anchor *= 4;
        }
    }

    int DecodePlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs,(float *)outputs[0],stream,batchSize);

        return 0;
    };

    PluginFieldCollection DecodePluginCreator::mFC{};
    std::vector<PluginField> DecodePluginCreator::mPluginAttributes;

    DecodePluginCreator::DecodePluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* DecodePluginCreator::getPluginName() const
    {
        return "Decode_TRT";
    }

    const char* DecodePluginCreator::getPluginVersion() const
    {
        return "1";
    }

    const PluginFieldCollection* DecodePluginCreator::getFieldNames()
    {
        return &mFC;
    }

    IPluginV2IOExt* DecodePluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        DecodePlugin* obj = new DecodePlugin();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* DecodePluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        // call PReluPlugin::destroy()
        DecodePlugin* obj = new DecodePlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

}
